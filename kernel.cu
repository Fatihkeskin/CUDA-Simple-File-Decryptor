#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>



//////////////////////////////////////////////////////////////////
//This Part is the parallel version of simple file decryptor
//////////////////////////////////////////////////////////////////
__global__ void CudaReadFile(char *a , int size){

	int pid = blockIdx.x*blockDim.x + threadIdx.x;
	if (pid < size)
		printf("%c", a[pid]);
	return;

}


///////////////////////////////////////////////////////////////////////////////////////////
//This Part(Half of the Main Function) is the serial version of simple file decryptor
///////////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char *argv[])
{

	//Determining the variables which are used
	FILE *file;
	char *special_message;
	char *buffer;
	int fileLen;
	int messageLen;
	


	//Opening the file
	file = fopen("encodedfile.txt", "rb");
	if (!file)
	{
		fprintf(stderr, "Unable to open file %s", "encodedfile.txt");
		return;
	}



	//Calc file length
	fseek(file, 0, SEEK_END);
	fileLen = ftell(file);
	fseek(file, 0, SEEK_SET);



	//Allocation
	messageLen = fileLen * 4;
	buffer = (char*)malloc(fileLen + 1);
	special_message = (char *)malloc(messageLen + 1);
	if (!buffer)
	{
		fprintf(stderr, "Memory error!");
		fclose(file);
		return;
	}


	//Read file contents into buffer
	fread(buffer, fileLen, 1, file);
	fclose(file);



	//Finding the message content
	char special_char = ',';
	int a;
	int special_counter = 0;
	for (a = 0; a < fileLen; a++){
		if (buffer[a] == special_char)
		{
			special_message[special_counter] = buffer[a + 1];
			special_counter++;
		}
	}



	//Writing the Special Message
	FILE *output_pointer = fopen("decoded.txt", "w");
	if (output_pointer == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}

	int r;
	for (r = 0; r < special_counter; r++)
	{
		fprintf(output_pointer, "%c", special_message[r]);
	}

	
	////////////////////////////////////////////////////////////////////////////////// Parallel directives
	char *gpuBuffer;
	char *gpuSpecialMessageBuffer;

	hipMalloc(&gpuBuffer, fileLen+1);
	hipMalloc(&gpuSpecialMessageBuffer, messageLen+1);

	hipMemcpy(gpuBuffer, buffer, fileLen*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(gpuSpecialMessageBuffer, special_message, fileLen*sizeof(char), hipMemcpyHostToDevice);

	//CudaReadFile << <60, 256 >> >(gpuBuffer, fileLen);

	//cudaMemcpy(gpuBuffer, gpuSpecialMessageBuffer, fileLen*sizeof(char), cudaMemcpyDeviceToHost);
	//cudaDeviceSynchronize();
	free(buffer);
	free(special_message);

	hipFree(gpuBuffer);
	hipFree(gpuSpecialMessageBuffer);

	return 0;

}//End of main
